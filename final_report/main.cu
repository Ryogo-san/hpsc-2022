
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <tuple>
#include <cmath>
#include <iterator>
#include <omp.h>
//#include <matplotlib-cpp/matplotlibcpp.h>
//namespace plt = matplotlibcpp;

const int M=1024;
const int nx=41;
const int ny=41;
const int nt=500;
const int nit=500;
const double dx=2./(double)(nx-1);
const double dy=2./(double)(ny-1);
const double dt=0.01;
const int rho=1;
const double nu=0.02;

__global__ void linspace(double *target,double min,double max,int n){
    double dx=(max-min)/(double)(n-1);
    int ii;
    ii=threadIdx.x;
    if(ii>=n)
        return;
    else if(ii==n-1)
        target[ii]=max;
    else
        target[ii]=(double)ii*dx+min;
}

__global__ void meshgrid(double *X, double *Y,double* vec1, double* vec2, int N){
    int jx=blockDim.x*blockIdx.x+threadIdx.x;
    int jy=blockDim.y*blockIdx.y+threadIdx.y;
    int i=nx*jy+jx;
    if(i>=N) return;
    X[i]=vec1[jx];
    Y[i]=vec2[jy];
}

__global__ void bBlock(double *b,double *u,double *v,int N){
    int jx=blockDim.x*blockIdx.x+threadIdx.x;
    int jy=blockDim.y*blockIdx.y+threadIdx.y;
    int i=nx*jy+jx;
    if(i>=N) return;
    if(jy>0&&jy<ny-1&&jx>0&&jx<nx-1){
        b[i]=rho*(1/dt*\
                ((u[i+1]-u[i-1])/(2*dx)+(v[i+nx]-v[i-nx])/(2*dy))-\
                std::pow(((u[i+1]-u[i-1])/(2*dx)),2)-2*((u[i+nx]-u[i-nx])/(2*dy)*\
                    (v[i+1]-v[i-1])/(2*dx)-std::pow(((v[i+nx]-v[i-nx])/(2*dy)),2)));
    }
}

__global__ void pBlock(double *p, double *pn, double *b,int N){
    int jx=blockDim.x*blockIdx.x+threadIdx.x;
    int jy=blockDim.y*blockIdx.y+threadIdx.y;
    int i=nx*jy+jx;
    if(i>=N) return;
    if(jy>0&&jy<ny-1&&jx>0&&jx<nx-1){
        p[i]=(std::pow(dy,2)*(pn[i+1]+pn[i-1])+\
                std::pow(dx,2)*(pn[i+nx]+pn[i-nx])-\
                b[i]*std::pow(dx,2)*std::pow(dy,2));
    }
}

__global__ void uvBlock(double *u,double *v,double *un,double *vn,double *p,int N){
    int jx=blockDim.x*blockIdx.x+threadIdx.x;
    int jy=blockDim.y*blockIdx.y+threadIdx.y;
    int i=nx*jy+jx;
    if(i>=N) return;
    if(jy>0&&jy<ny-1&&jx>0&&jx<nx-1){
        u[i]=un[i]-un[i]*dt/dx*(un[i]-un[i-1])\
             -un[i]*dt/dy*(un[i]-un[i-nx])\
             -dt/(2*rho*dx)*(p[i+1]-p[i-1])\
             +nu*dt/std::pow(dx,2)*(un[i+1]-2*un[i]+un[i-1])\
             +nu*dt/std::pow(dy,2)*(un[i+nx]-2*un[i]+un[i-nx]);
        v[i]=vn[i]-vn[i]*dt/dx*(vn[i]-vn[i-1])\
             -vn[i]*dt/dy*(vn[i]-vn[i-nx])\
             -dt/(2*rho*dx)*(p[i+nx]-p[i-nx])\
             +nu*dt/std::pow(dx,2)*(vn[i+1]-2*vn[i]+vn[i-1])\
             +nu*dt/std::pow(dy,2)*(vn[i+nx]-2*vn[i]+vn[i-nx]);
    }
}
__global__ void copy(double *dst, double *src, int N){
    int jx=blockDim.x*blockIdx.x+threadIdx.x;
    int jy=blockDim.y*blockIdx.y+threadIdx.y;
    int i=nx*jy+jx;
    if(i>=N) return;
    dst[i]=src[i];
}

int main(){
    double* x=NULL;
    double* y=NULL;
    double* u=NULL;
    double* v=NULL;
    double* p=NULL;
    double* b=NULL;
    double* X=NULL;
    double* Y=NULL;
    const int N=nx*ny;
    hipMallocManaged(&x,nx*sizeof(double));
    hipMallocManaged(&y,ny*sizeof(double));
    linspace<<<(N+M-1)/M,M>>>(x,0.,2.,nx);
    hipDeviceSynchronize();
    linspace<<<(N+M-1)/M,M>>>(y,0.,2.,ny);
    hipDeviceSynchronize();
    // zeros
    hipMallocManaged(&u,ny*nx*sizeof(double));
    hipMemset(u,0,sizeof(double)*ny*nx);
    hipMallocManaged(&v,ny*nx*sizeof(double));
    hipMemset(v,0,sizeof(double)*ny*nx);
    hipMallocManaged(&p,ny*nx*sizeof(double));
    hipMemset(p,0,sizeof(double)*ny*nx);
    hipMallocManaged(&b,ny*nx*sizeof(double));
    hipMemset(b,0,sizeof(double)*ny*nx);

    /*
    for(int i=0;i<ny;i++){
        for(int j=0;j<nx;j++)
            std::cout<<u[i*ny+j]<<" ";
        std::cout<<std::endl;
    }
    */

    hipMallocManaged(&X,ny*nx*sizeof(double));
    hipMallocManaged(&Y,ny*nx*sizeof(double));

    // meshgrid
    meshgrid<<<(N+M-1)/M,M>>>(X,Y,x,y,ny*nx);
    hipDeviceSynchronize();

    for(int n=0;n<nt;n++){
        bBlock<<<(N+M-1)/M,M>>>(b,u,v,ny*nx);
        hipDeviceSynchronize();
        
        for(int it=0;it<nit;it++){
            double* pn=NULL;
            hipMallocManaged(&pn,ny*nx*sizeof(double));
            copy<<<(N+M-1)/M,M>>>(pn,p,ny*nx);
            hipDeviceSynchronize();
            
            pBlock<<<(N+M-1)/M,M>>>(p, pn, b, ny*nx);
            hipDeviceSynchronize();

#pragma omp parallel for
            for(int i=0;i<ny;i++) p[i*nx+nx-1]=p[i*nx+nx-2];
#pragma omp parallel for
            for(int j=0;j<nx;j++) p[0+j]=p[nx+j];
#pragma omp parallel for
            for(int i=0;i<ny;i++) p[i*nx]=p[i*nx+1];
#pragma omp parallel for
            for(int j=0;j<nx;j++) p[(ny-1)*nx+j]=0;
     
            hipFree(pn);
        }

        double* un=NULL;
        hipMallocManaged(&un,ny*nx*sizeof(double));
        copy<<<(N+M-1)/M,M>>>(un,u,ny*nx);
        hipDeviceSynchronize();
        double* vn=NULL;;
        hipMallocManaged(&vn,ny*nx*sizeof(double));
        copy<<<(N+M-1)/M,M>>>(vn,v,ny*nx);
        hipDeviceSynchronize();

        uvBlock<<<(N+M-1)/M,M>>>(u,v,un,vn,p,ny*nx);
        hipDeviceSynchronize();

#pragma omp parallel for
        for(int j=0;j<nx;j++) u[j]=0;
#pragma omp parallel for
        for(int i=0;i<ny;i++) u[i*nx]=0;
#pragma omp parallel for
        for(int i=0;i<ny;i++) u[i*nx+nx-1]=0;
#pragma omp parallel for
        for(int j=0;j<nx;j++) u[(ny-1)*nx+j]=1;
#pragma omp parallel for
        for(int j=0;j<nx;j++) v[j]=0;
#pragma omp parallel for
        for(int j=0;j<nx;j++) v[(ny-1)*nx+j]=0;
#pragma omp parallel for
        for(int i=0;i<ny;i++) v[i*nx]=0;
#pragma omp parallel for
        for(int i=0;i<ny;i++) v[i*nx+nx-1]=0;

        //plt::contourf(X,Y,p,0.5);
        //plt::quiver(X,Y,u,v);
        //plt::pause(0.01);
        //plt::clf();
        hipFree(un);
        hipFree(vn);
    }
    //plt::show();
    
    hipFree(x);
    hipFree(y);
    hipFree(u);
    hipFree(v);
    hipFree(p);
    hipFree(b);
    hipFree(X);
    hipFree(Y);
    return 0;
}
